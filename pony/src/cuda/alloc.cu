#include <cstdlib>
#include <stdexcept>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <fmt/core.h>

#include "alloc.cuh"

namespace dory::cuda {
GpuCudaAllocator::GpuCudaAllocator(size_t bytes) : ArenaAllocator(bytes) {
  hipError_t result = hipMalloc(&data, sz);

  if (result != hipSuccess) {
    throw std::runtime_error(fmt::format("Could not allocate CUDA memory: {}\n",
                                         hipGetErrorString(result)));
  }

  p = data;
}

GpuCudaAllocator::~GpuCudaAllocator() noexcept(false) {
  if (data) {
    hipError_t result = hipFree(data);

    if (result != hipSuccess) {
      throw std::runtime_error(fmt::format("Could not free CUDA memory: {}\n",
                                           hipGetErrorString(result)));
    }
  }
}

GpuNormalAllocator::GpuNormalAllocator(size_t bytes) : ArenaAllocator(bytes) {
  auto ptr = std::malloc(sz);

  if (!ptr) {
    throw std::runtime_error("Could not allocate CUDA memory using malloc\n");
  }

  data = reinterpret_cast<decltype(data)>(ptr);
  p = data;
}

GpuNormalAllocator::~GpuNormalAllocator() noexcept(false) { std::free(data); }

HostCudaAllocator::HostCudaAllocator(size_t bytes) : ArenaAllocator(bytes) {
  hipError_t result = hipHostMalloc(&data, sz);

  if (result != hipSuccess) {
    throw std::runtime_error(fmt::format("Could not allocate HOST memory: {}\n",
                                         hipGetErrorString(result)));
  }

  p = data;
}

HostCudaAllocator::~HostCudaAllocator() noexcept(false) {
  if (data) {
    hipError_t result = hipHostFree(data);

    if (result != hipSuccess) {
      throw std::runtime_error(fmt::format("Could not free HOST memory: {}\n",
                                           hipGetErrorString(result)));
    }
  }
}

HostNormalAllocator::HostNormalAllocator(size_t bytes) : ArenaAllocator(bytes) {
  auto ptr = std::malloc(sz);

  if (!ptr) {
    throw std::runtime_error("Could not allocate HOST memory using malloc\n");
  }

  data = reinterpret_cast<decltype(data)>(ptr);
  p = data;
}

HostNormalAllocator::~HostNormalAllocator() noexcept(false) { std::free(data); }

}  // namespace dory::cuda
