#include "hip/hip_runtime.h"
#include <cstdio>
#include <stdexcept>

#include <fmt/core.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "check.cuh"
#include "util.cuh"

__global__ void cuda_check_kernel_invocation(bool print) {
  if (print) {
    printf("GPU kernel invocation works!\n");
  }
}

namespace dory::cuda {
bool works(bool print) {
  hipError_t err;

  cuda_check_kernel_invocation<<<1, 1>>>(print);

  err = hipPeekAtLastError();
  gpuAssert(err, __FILE__, __LINE__, false);
  if (err != hipSuccess) {
    return false;
  }

  err = hipDeviceSynchronize();
  gpuAssert(err, __FILE__, __LINE__, false);
  if (err != hipSuccess) {
    return false;
  }

  return true;
}

bool have_gpu() {
  int deviceCount = 0;
  hipError_t error;

  error = hipInit(0);
  if (error != hipSuccess) {
    return false;
  }

  error = hipGetDeviceCount(&deviceCount);
  if (error != hipSuccess) {
    throw std::runtime_error(
        fmt::format("Failed to query the number of CUDA devices ({})\n",
                    static_cast<int>(error)));
  }

  return deviceCount > 0;
}
}  // namespace dory::cuda
